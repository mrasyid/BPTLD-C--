#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 17.10.2011
 *      Author: SYavorovsky@varnasoftware.com
 */

#include "kernels.cuh"


__device__  double bbOverlap(int *bb1, int *bb2) {
    // Check whether the bounding-boxes overlap at all
    if (bb1[0] > bb2[0] + bb2[2]) {
        return 0;
    }
    else if (bb1[1] > bb2[1] + bb2[3]) {
        return 0;
    }
    else if (bb2[0] > bb1[0] + bb1[2]) {
        return 0;
    }
    else if (bb2[1] > bb1[1] + bb1[3]) {
        return 0;
    }


    // If we got this far, the bounding-boxes overlap
    double overlapWidth = min(bb1[0] + bb1[2], bb2[0] + bb2[2]) - max(bb1[0], bb2[0]);
    double overlapHeight = min(bb1[1] + bb1[3], bb2[1] + bb2[3]) - max(bb1[1], bb2[1]);
    double overlapArea = overlapWidth * overlapHeight;
    double bb1Area = bb1[2] * bb1[3];
    double bb2Area = bb2[2] * bb2[3];

    return overlapArea / (bb1Area + bb2Area - overlapArea);
}


__device__ int sumRect(int *IIdata, int width, int height, int x, int y, int w, int h)
{
	int retval=0;

    if (x >= 0 && w > 0 && x + w < width && y >= 0 && h > 0 && y + h < height) {

    	int dx=IIdata[x+y*(width)];
    	int dy=IIdata[(x+w)+(y+h)*(width)];
    	int dw=IIdata[(x+w)+y*(width)];
    	int dh=IIdata[x+(y+h)*(width)];

    	retval=dx+dy-dw-dh;

         // IIdata[x][y] + IIdata[x + w][y + h] - IIdata[x + w][y] - IIdata[x][y + h];

     } else {
    	 printf("ERROR: SUM RECT OUT OF BOUNDS! (%d, %d, %d, %d)\n", x, y, w, h);
     }
        return retval;
}


__device__ int getWarpXY(int *IIdata, int width, int height, int wx,int wy,int ww,int wh,int*bb,float *m)
{
    int ox = -(int)((float)(ww) * 0.5);
    int oy = -(int)((float)(wh) * 0.5);
    int cx = (int)(bb[0] - ox);
    int cy = (int)(bb[1] - oy);

	int x=ox+wx;
	int y=oy+wy;
    int xp = (int)(m[0] * (float)x + m[1] * (float)y + cx);
    int yp = (int)(m[2] * (float)x + m[3] * (float)y + cy);

    // Limit pixels to those in the given bounding-box
    xp = max(min(xp, bb[0] + ww), bb[0]);
    yp = max(min(yp, bb[1] + wh), bb[1]);

    return IIdata[xp + yp*width];
}

__device__ int sumRectWarp(int *IIdata, int width, int height, int x, int y, int w, int h, int *bb, float *m)
{
	int retval=0;

    if (x >= 0 && w > 0 && x + w < width && y >= 0 && h > 0 && y + h < height) {

    	int dx=getWarpXY(IIdata,width, height, x,y,w,h,bb,m);
    	int dy=getWarpXY(IIdata,width, height, x+w,y+h,w,h,bb,m);
    	int dw=getWarpXY(IIdata,width, height, x+w,y,w,h,bb,m);
    	int dh=getWarpXY(IIdata,width, height, x,y+h,w,h,bb,m);

    	retval=dx+dy-dw-dh;

         // IIdata[x][y] + IIdata[x + w][y + h] - IIdata[x + w][y] - IIdata[x][y + h];

     } else {
    	 printf("ERROR: SUM RECT OUT OF WARP BOUNDS! (%d, %d, %d, %d)\n", x, y, w, h);
     }
        return retval;
}


/*
 * in_ferns  r/w лес деревьев
 * IIdata Integral Image
 * w,h size of Integral Image
 * patch[]={x,y,w,h,class} - Patch parametrs
 */

__global__ void train_kernel(struct one_fern *in_ferns,int *IIdata, int width, int height, int p_idx2, int* patch, int* tbb )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y; //y - this a p_idx = 0..6*30*30

	int leafIdx=0;
	if(idx>=TOTAL_FERNS)return;
	//найдем индекс ветки
	if(idy>=p_idx2)return;
	int p_idx=idy;

	int patchX=patch[p_idx*5+0];
	int patchY=patch[p_idx*5+1];
	int patchW=patch[p_idx*5+2];
	int patchH=patch[p_idx*5+3];
	if(!(patchX+patchY+patchW+patchH))
	{
		printf("error in %d",p_idx);
		return;
	}
    // Clamp x and y values between 0 and width and height respectively
    patchX = max(min(patchX, width - 2), 0);
    patchY = max(min(patchY, height - 2), 0);

    // Limit width and height values to (width - patchX) and (height - patchY)
    // respectively
    patchW = min(patchW, width - patchX);
    patchH = min(patchH, height - patchY);

    // Apply all tests to find the leaf index this patch falls into
    int leaf = 0;
    int test=0;

    /*for (int i = 0; i < nodeCount; i++) {
        leaf = leaf | (nodes[i]->test(image, patchX, patchY, patchW, patchH) << i * (int)POWER);
    }*/
    int i;
	for(i=0;i<TOTAL_NODES;i++)
	{

	    int x = (int)(in_ferns[idx].nodes[i][0] * (float)patchW) + patchX;
	    int y = (int)(in_ferns[idx].nodes[i][1] * (float)patchH) + patchY;
	    int w = (int)(in_ferns[idx].nodes[i][2] * (float)patchW * 0.5f);
	    int h = (int)(in_ferns[idx].nodes[i][3] * (float)patchH * 0.5f);

	    // Compare the various halfs of the feature on the patch
	    int left,right,top,bottom;

	    left = sumRect(IIdata, width, height, x, y, w, h * 2);
	    right = sumRect(IIdata, width, height, x + w, y, w, h * 2);
	    top = sumRect(IIdata, width, height, x, y, w * 2, h);
	    bottom = sumRect(IIdata, width, height, x, y + h, w * 2, h);

	    if (left > right) {
	        if (top > bottom) {
	        	test=0;
	        }
	        else {
	        	test=1;
	        }
	    }
	    else {
	        if (top > bottom) {
	        	test=2;
	        }
	        else {
	        	test=3;
	        }
	    }


		leaf = leaf | (test << (i * (int)2));

	}

	int is_positive=patch[p_idx*5+4];

	if(tbb!=NULL)
	{
		if (bbOverlap(tbb, &patch[p_idx*5]) >= MIN_LEARNING_OVERLAP)
			is_positive=1;
	}

	//if(p_idx2>1)printf(" %d; ",is_positive);

    if (is_positive == 0) {
    	atomicAdd(&in_ferns[idx].n[leaf],1);
    }
    else {
    	atomicAdd(&in_ferns[idx].p[leaf],1);
    }
   // int p=in_ferns[idx].p[leaf];
   // int n=in_ferns[idx].n[leaf];
    // Compute the posterior likelihood of a positive class for this leaf
  /*  if (p > 0) {

    	in_ferns[idx].posteriors[leaf] = (float)p / (float)(p + n);
    }*/

}


__global__ void patcher_kernel(int count, int incX, int minX, int incY, int minY,int currentWidth, int currentHeight, int* patch, int s)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if(idx>=count || idy>=count)return;

	int p_len=idy+idx*count+s*count*count; //int p_len=idy+idx*county+s*countx*county;
	int x=idx*incX+minX;
	int y=idy*incY+minY;

	patch[(p_len)*5]=x;
	patch[(p_len)*5+1]=y;
	patch[(p_len)*5+2]=currentWidth;
	patch[(p_len)*5+3]=currentHeight;
}

__global__ void classify_kernel(struct one_fern *in_ferns,int *IIdata, int width, int height, int p_idx2, int* patch, float *ret )
{


	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y; //y - this a p_idx = 0..6*30*30

	int leafIdx=0;

	if(idx>=TOTAL_FERNS)return;

	if(idy>=p_idx2)return;
	int p_idx=idy;
	//if(p_idx2>1)printf("ok %d %d; ",idy,p_idx2);
	int patchX=patch[p_idx*5+0];
	int patchY=patch[p_idx*5+1];
	int patchW=patch[p_idx*5+2];
	int patchH=patch[p_idx*5+3];
	if(!(patchX+patchY+patchW+patchH))
	{
		printf("error in %d",p_idx);
		return;
	}
    // Clamp x and y values between 0 and width and height respectively
    patchX = max(min(patchX, width - 2), 0);
    patchY = max(min(patchY, height - 2), 0);

    // Limit width and height values to (width - patchX) and (height - patchY)
    // respectively
    patchW = min(patchW, width - patchX);
    patchH = min(patchH, height - patchY);

    // Apply all tests to find the leaf index this patch falls into
    int leaf = 0;
    int test=0;

    /*for (int i = 0; i < nodeCount; i++) {
        leaf = leaf | (nodes[i]->test(image, patchX, patchY, patchW, patchH) << i * (int)POWER);
    }*/
    int i;
	for(i=0;i<TOTAL_NODES;i++)
	{

	    int x = (int)(in_ferns[idx].nodes[i][0] * (float)patchW) + patchX;
	    int y = (int)(in_ferns[idx].nodes[i][1] * (float)patchH) + patchY;
	    int w = (int)(in_ferns[idx].nodes[i][2] * (float)patchW * 0.5f);
	    int h = (int)(in_ferns[idx].nodes[i][3] * (float)patchH * 0.5f);

	    // Compare the various halfs of the feature on the patch
	    int left,right,top,bottom;

	    left = sumRect(IIdata, width, height, x, y, w, h * 2);
	    right = sumRect(IIdata, width, height, x + w, y, w, h * 2);
	    top = sumRect(IIdata, width, height, x, y, w * 2, h);
	    bottom = sumRect(IIdata, width, height, x, y + h, w * 2, h);

	    if (left > right) {
	        if (top > bottom) {
	        	test=0;
	        }
	        else {
	        	test=1;
	        }
	    }
	    else {
	        if (top > bottom) {
	        	test=2;
	        }
	        else {
	        	test=3;
	        }
	    }


		leaf = leaf | (test << (i * (int)2));

	}
	//printf("Hello thread %d, f=%d\n", threadIdx.x, leaf);
	//wait for all treads
	//ret[idx+p_idx*TOTAL_FERNS]=in_ferns[idx].posteriors[leaf];
    int p=in_ferns[idx].p[leaf];
    int n=in_ferns[idx].n[leaf];
    if(p || n)
	atomicAdd(&ret[p_idx],(float)p / (float)(p + n));
	//atomicAdd(&ret[p_idx],in_ferns[idx].posteriors[leaf]);

	__syncthreads();
	//
	/*if(idx==0)
	{
		*ret=c_retval[0];//(float)TOTAL_FERNS;
	}*/


}

__global__ void train_kernel_warp(struct one_fern *in_ferns,int *IIdata, int width, int height, int* patch, int* bb, float *m )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int leafIdx=0;
	if(idx>=TOTAL_FERNS)return;
	//найдем индекс ветки
	//int p_idx=0;

	int patchX=patch[0];
	int patchY=patch[1];
	int patchW=patch[2];
	int patchH=patch[3];

    // Clamp x and y values between 0 and width and height respectively
    // Clamp x and y values between 0 and width and height respectively
    patchX = max(min(patchX, width - 2), 0);
    patchY = max(min(patchY, height - 2), 0);

    // Limit width and height values to (width - patchX) and (height - patchY)
    // respectively
    patchW = min(patchW, width - patchX);
    patchH = min(patchH, height - patchY);
    //__syncthreads ();
    // Apply all tests to find the leaf index this patch falls into
    int leaf = 0;
    int test=0;

    /*for (int i = 0; i < nodeCount; i++) {
        leaf = leaf | (nodes[i]->test(image, patchX, patchY, patchW, patchH) << i * (int)POWER);
    }*/
    int i;
	for(i=0;i<TOTAL_NODES;i++)
	{

	    int x = (int)(in_ferns[idx].nodes[i][0] * (float)patchW) + patchX;
	    int y = (int)(in_ferns[idx].nodes[i][1] * (float)patchH) + patchY;
	    int w = (int)(in_ferns[idx].nodes[i][2] * (float)patchW * 0.5f);
	    int h = (int)(in_ferns[idx].nodes[i][3] * (float)patchH * 0.5f);


	    // Compare the various halfs of the feature on the patch
	    int left,right,top,bottom;
	    left = sumRectWarp(IIdata, width, height, x, y, w, h * 2,bb,m);
	    right = sumRectWarp(IIdata, width, height, x + w, y, w, h * 2,bb,m);
	    top = sumRectWarp(IIdata, width, height, x, y, w * 2, h,bb,m);
	    bottom = sumRectWarp(IIdata, width, height, x, y + h, w * 2, h,bb,m);


	    if (left > right) {
	        if (top > bottom) {
	        	test=0;
	        }
	        else {
	        	test=1;
	        }
	    }
	    else {
	        if (top > bottom) {
	        	test=2;
	        }
	        else {
	        	test=3;
	        }
	    }


		leaf = leaf | (test << (i * (int)2));

	}

/*
	int is_positive=patch[p_idx*5+4];
	if(tbb!=NULL)
	{
		if (bbOverlap(tbb, &patch[p_idx*5]) >= MIN_LEARNING_OVERLAP)
			is_positive=1;
	}


    if (is_positive == 0) {
    	atomicAdd(&in_ferns[idx].n[leaf],1);
    }
    else {
    	atomicAdd(&in_ferns[idx].p[leaf],1);
    }
*/

    if (patch[4] == 0) {
    	atomicAdd(&in_ferns[idx].n[leaf],1);
    }
    else {
    	atomicAdd(&in_ferns[idx].p[leaf],1);
    }
/*  int p=in_ferns[idx].p[leaf];
    int n=in_ferns[idx].n[leaf];
    // Compute the posterior likelihood of a positive class for this leaf
    if (p > 0) {

    	in_ferns[idx].posteriors[leaf] = (float)p / (float)(p + n);
    }
*/
}

